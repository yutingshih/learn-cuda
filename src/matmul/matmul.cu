#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "matmul.h"

__global__
void _matMul(float *M, float *N, float *P, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= size || j >= size) return;

    float sum = 0.0;
    for (int k = 0; k < size; ++k) {
        sum += M[i * size + k] * N[k * size + j];
    }
    P[i * size + j] = sum;
}

void matMul(float *M, float *N, float *P, int size) {
    int nbytes = size * size * sizeof(float);
    float *_M, *_N, *_P;
    hipMalloc(&_M, nbytes);
    hipMalloc(&_N, nbytes);
    hipMalloc(&_P, nbytes);

    hipMemcpy(_M, M, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(_N, N, nbytes, hipMemcpyHostToDevice);

    int tileSize = 2, tileNum = 1 + (size - 1) / tileSize;
    dim3 dimGrid(tileNum, tileNum, 1);
    dim3 dimBlock(tileSize, tileSize, 1);
    _matMul<<<dimGrid, dimBlock>>>(_M, _N, _P, size);

    hipMemcpy(P, _P, nbytes, hipMemcpyDeviceToHost);

    hipFree(_M);
    hipFree(_N);
    hipFree(_P);
}

int main() {
    const int size = 4;
    int nbytes = size * size * sizeof(float);
    float *M = (float *)malloc(nbytes);
    float *N = (float *)malloc(nbytes);
    float *P = (float *)malloc(nbytes);

    matInit(M, size);
    matInit(N, size);
    matMul(M, N, P, size);
    matShow(M, size);
    matShow(N, size);
    matShow(P, size);

    free(M), free(N), free(P);
    return 0;
}
