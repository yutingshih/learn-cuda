#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void randomNumber(float *x, int n) {
    hiprandState state;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        uint64_t seed = (uint64_t) clock() + (uint64_t) tid * 1234;
        hiprand_init(seed, tid, 0, &state);
        x[tid] = hiprand_uniform(&state);
    }
}

void printArray(float *a, int n) {
    for (int i = 0; i < n; i++)
        std::cout << a[i] << " ";
    std::cout << std::endl;
}

int main() {
    int nElem = 10;
    int nBytes = sizeof(float) * nElem;
    float *_x;
    hipMalloc(&_x, nBytes);

    randomNumber<<<1, 10>>>(_x, nElem);

    float *x = (float *)malloc(nBytes);
    hipMemcpy(x, _x, nBytes, hipMemcpyDeviceToHost);
    printArray(x, nElem);
    hipFree(_x);
    free(x);
    return 0;
}
